
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <bits/stdc++.h>

using namespace std;

__global__ void matrixMultiply(int *a, int *b, int *c, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += a[row * N + k] * b[k * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main()
{
    int N = 4; 

    int *a, *b, *c; 
    int *d_a, *d_b, *d_c; 

    int matrixSize = N * N * sizeof(int);

    a = (int*)malloc(matrixSize);
    b = (int*)malloc(matrixSize);
    c = (int*)malloc(matrixSize);

    for (int i = 0; i < N * N; ++i) {
        a[i] = rand()%1000;
        b[i] = rand()%1000;
    }


    hipMalloc((void**)&d_a, matrixSize);
    hipMalloc((void**)&d_b, matrixSize);
    hipMalloc((void**)&d_c, matrixSize);

    hipMemcpy(d_a, a, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, matrixSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(2, 2);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);


    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, matrixSize, hipMemcpyDeviceToHost);
    printf("Multiplied Number: ");
    for (int i = 0; i < N * N; ++i) {
        std::cout << c[i] << " ";
        if ((i + 1) % N == 0)
            cout<<endl;
    }

    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;}

